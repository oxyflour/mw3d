#include <napi.h>
#include <hip/hip_runtime.h>

Napi::Value Test(const Napi::CallbackInfo &info) {
    return info.Env().Undefined();
}

Napi::Object Init(Napi::Env env, Napi::Object exports) {
    exports.Set("test", Napi::Function::New(env, Test));
    return exports;
}

NODE_API_MODULE(binding, Init)
